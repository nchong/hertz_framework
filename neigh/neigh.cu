#include "hip/hip_runtime.h"
/*
 * Neighbor list decomposition of the hertz pairwise kernel. 
 */

#define NSLOT 32
//#define KERNEL_PRINT    //< debug printing in kernel

#ifdef GPU_TIMER
  #include "cuda_timer.h"
#elif POSIX_TIMER
  #include "posix_timer.h"
#else
  #include "simple_timer.h"
#endif

#include "check_result_vector.h"
#include "cuda_common.h"
#include "framework.h"
#include "hertz_constants.h"
#include "particle.h"
#include <sstream>

using namespace std;

// --------------------------------------------------------------------------
// DEVICE KERNEL
// --------------------------------------------------------------------------

__device__ void pair_interaction(
  //inputs
    double *xi, double *xj,           //position
    double *vi, double *vj,           //velocity
    double *omegai, double *omegaj,   //rotational velocity
    double radi, double radj,         //radius
    double massi, double massj,       //mass
    int typei, int typej,             //type
  //inouts
    double *shear,
    double *torque,
    double *force) {

  // del is the vector from j to i
  double delx = xi[0] - xj[0];
  double dely = xi[1] - xj[1];
  double delz = xi[2] - xj[2];

  double rsq = delx*delx + dely*dely + delz*delz;
  double radsum = radi + radj;
  if (rsq >= radsum*radsum) {
    //unset non-touching atoms
    shear[0] = 0.0;
    shear[1] = 0.0;
    shear[2] = 0.0;
  } else {
    //distance between centres of atoms i and j
    //or, magnitude of del vector
    double r = sqrt(rsq);
    double rinv = 1.0/r;
    double rsqinv = 1.0/rsq;

    // relative translational velocity
    double vr1 = vi[0] - vj[0];
    double vr2 = vi[1] - vj[1];
    double vr3 = vi[2] - vj[2];

    // normal component
    double vnnr = vr1*delx + vr2*dely + vr3*delz;
    double vn1 = delx*vnnr * rsqinv;
    double vn2 = dely*vnnr * rsqinv;
    double vn3 = delz*vnnr * rsqinv;

    // tangential component
    double vt1 = vr1 - vn1;
    double vt2 = vr2 - vn2;
    double vt3 = vr3 - vn3;

    // relative rotational velocity
    double wr1 = (radi*omegai[0] + radj*omegaj[0]) * rinv;
    double wr2 = (radi*omegai[1] + radj*omegaj[1]) * rinv;
    double wr3 = (radi*omegai[2] + radj*omegaj[2]) * rinv;

    // normal forces = Hookian contact + normal velocity damping
    double meff = massi*massj/(massi+massj);
    //not-implemented: freeze_group_bit

    double deltan = radsum-r;

    //derive contact model parameters (inlined)
    //Yeff, Geff, betaeff, coeffFrict are lookup tables
    double reff = radi * radj / (radi + radj);
    double sqrtval = sqrt(reff * deltan);
    double Sn = 2.    * d_yeff * sqrtval;
    double St = 8.    * d_geff * sqrtval;
    double kn = 4./3. * d_yeff * sqrtval;
    double kt = St;
    double gamman=-2.*sqrtFiveOverSix*d_betaeff*sqrt(Sn*meff);
    double gammat=-2.*sqrtFiveOverSix*d_betaeff*sqrt(St*meff);
    double xmu=d_coeffFrict;
    //not-implemented if (dampflag == 0) gammat = 0;
    kn /= d_nktv2p;
    kt /= d_nktv2p;

    double damp = gamman*vnnr*rsqinv;
	  double ccel = kn*(radsum-r)*rinv - damp;

    //not-implemented cohesionflag

    // relative velocities
    double vtr1 = vt1 - (delz*wr2-dely*wr3);
    double vtr2 = vt2 - (delx*wr3-delz*wr1);
    double vtr3 = vt3 - (dely*wr1-delx*wr2);

    // shear history effects
    shear[0] += vtr1 * d_dt;
    shear[1] += vtr2 * d_dt;
    shear[2] += vtr3 * d_dt;

    // rotate shear displacements
    double rsht = shear[0]*delx + shear[1]*dely + shear[2]*delz;
    rsht *= rsqinv;

    shear[0] -= rsht*delx;
    shear[1] -= rsht*dely;
    shear[2] -= rsht*delz;

    // tangential forces = shear + tangential velocity damping
    double fs1 = - (kt*shear[0] + gammat*vtr1);
    double fs2 = - (kt*shear[1] + gammat*vtr2);
    double fs3 = - (kt*shear[2] + gammat*vtr3);

    // rescale frictional displacements and forces if needed
    double fs = sqrt(fs1*fs1 + fs2*fs2 + fs3*fs3);
    double fn = xmu * fabs(ccel*r);
    double shrmag = 0;
    if (fs > fn) {
      shrmag = sqrt(shear[0]*shear[0] +
                    shear[1]*shear[1] +
                    shear[2]*shear[2]);
      if (shrmag != 0.0) {
        shear[0] = (fn/fs) * (shear[0] + gammat*vtr1/kt) - gammat*vtr1/kt;
        shear[1] = (fn/fs) * (shear[1] + gammat*vtr2/kt) - gammat*vtr2/kt;
        shear[2] = (fn/fs) * (shear[2] + gammat*vtr3/kt) - gammat*vtr3/kt;
        fs1 *= fn/fs;
        fs2 *= fn/fs;
        fs3 *= fn/fs;
      } else {
        fs1 = fs2 = fs3 = 0.0;
      }
    }

    double fx = delx*ccel + fs1;
    double fy = dely*ccel + fs2;
    double fz = delz*ccel + fs3;

    double tor1 = rinv * (dely*fs3 - delz*fs2);
    double tor2 = rinv * (delz*fs1 - delx*fs3);
    double tor3 = rinv * (delx*fs2 - dely*fs1);

    // this is what we've been working up to!
    force[0] += fx;
    force[1] += fy;
    force[2] += fz;

    torque[0] -= radi*tor1;
    torque[1] -= radi*tor2;
    torque[2] -= radi*tor3;
  }
}

__global__ void compute_kernel_tpa(
  int nparticles,   
  struct particle *particle_aos,
  int *numneigh,
  struct particle *neigh,
  double3 *shear,
  double *force,
  double *torque) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nparticles && numneigh[idx] > 0) {
    double xi[3]; double xj[3];
    double vi[3]; double vj[3];
    double omegai[3]; double omegaj[3];
    double radiusi; double radiusj;
    double massi; double massj;
    int    typei; int typej;
    double shearij[3] = {0,0,0};
    double force_deltai[3] = {0,0,0};
    double torque_deltai[3] = {0,0,0};

    xi[0]     = particle_aos[idx].x[0];
    xi[1]     = particle_aos[idx].x[1];
    xi[2]     = particle_aos[idx].x[2];
    vi[0]     = particle_aos[idx].v[0];
    vi[1]     = particle_aos[idx].v[1];
    vi[2]     = particle_aos[idx].v[2];
    omegai[0] = particle_aos[idx].omega[0];
    omegai[1] = particle_aos[idx].omega[1];
    omegai[2] = particle_aos[idx].omega[2];
    radiusi   = particle_aos[idx].radius;
    massi     = particle_aos[idx].mass;
    typei     = particle_aos[idx].type;

    for (int jj=0; jj<numneigh[idx]; jj++) {
      int neigh_idx = (idx*NSLOT)+jj;
      //int j   = neigh[neigh_idx].idx;
      xj[0]     = neigh[neigh_idx].x[0];
      xj[1]     = neigh[neigh_idx].x[1];
      xj[2]     = neigh[neigh_idx].x[2];
      vj[0]     = neigh[neigh_idx].v[0];
      vj[1]     = neigh[neigh_idx].v[1];
      vj[2]     = neigh[neigh_idx].v[2];
      omegaj[0] = neigh[neigh_idx].omega[0];
      omegaj[1] = neigh[neigh_idx].omega[1];
      omegaj[2] = neigh[neigh_idx].omega[2];
      radiusj   = neigh[neigh_idx].radius;
      massj     = neigh[neigh_idx].mass;
      typej     = neigh[neigh_idx].type;

      shearij[0] = shear[neigh_idx].x;
      shearij[1] = shear[neigh_idx].y;
      shearij[2] = shear[neigh_idx].z;

      pair_interaction(
        xi, xj,
        vi, vj,
        omegai, omegaj,
        radiusi, radiusj,
        massi, massj,
        typei, typej,
        shearij, torque_deltai, force_deltai);

      shear[neigh_idx].x = shearij[0];
      shear[neigh_idx].y = shearij[1];
      shear[neigh_idx].z = shearij[2];
    }
    force[(idx*3)  ] += force_deltai[0];
    force[(idx*3)+1] += force_deltai[1];
    force[(idx*3)+2] += force_deltai[2];

    torque[(idx*3)  ] += torque_deltai[0];
    torque[(idx*3)+1] += torque_deltai[1];
    torque[(idx*3)+2] += torque_deltai[2];
  }
}

// --------------------------------------------------------------------------
// RUN
// --------------------------------------------------------------------------

void insert_particle(struct params *input, struct particle *aos, int id, int n) {
  assert(n < input->nnode);
  aos[id].idx      = n;
  aos[id].x[0]     = input->x[(n*3)  ];
  aos[id].x[1]     = input->x[(n*3)+1];
  aos[id].x[2]     = input->x[(n*3)+2];
  aos[id].v[0]     = input->v[(n*3)  ];
  aos[id].v[1]     = input->v[(n*3)+1];
  aos[id].v[2]     = input->v[(n*3)+2];
  aos[id].omega[0] = input->omega[(n*3)  ];
  aos[id].omega[1] = input->omega[(n*3)+1];
  aos[id].omega[2] = input->omega[(n*3)+2];
  aos[id].radius   = input->radius[n];
  aos[id].mass     = input->mass[n];
  aos[id].type     = input->type[n];
}

void build_particle_aos(struct params *input, struct particle *&d_particle_aos) {
  struct particle *aos = new particle[input->nnode];
  for (int n=0; n<input->nnode; n++) {
    insert_particle(input, aos, n, n);
  }
  const int aos_size = input->nnode*sizeof(struct particle);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_particle_aos, aos_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_particle_aos, aos, aos_size, hipMemcpyHostToDevice));
}

void build_neighbor_list(
  int nslot,
  struct params *input, 
  int *&d_numneigh, struct particle *&d_neigh, double3 *&d_shear) {

  int *numneigh = new int[input->nnode*nslot];
  struct particle *neigh = new particle[input->nnode*nslot];
  double3 *shear = new double3[input->nnode*nslot];

  for (int i=0; i<input->nnode*nslot; i++) {
    numneigh[i] = 0;
  }
  for (int e=0; e<input->nedge; e++) {
    int i = input->edge[(e*2)  ];
    int j = input->edge[(e*2)+1];

    assert(numneigh[i] < nslot);
    int idx = (i*nslot) + numneigh[i];
    insert_particle(input, neigh, idx, j);
    shear[idx].x = input->shear[(e*3)  ];
    shear[idx].y = input->shear[(e*3)+1];
    shear[idx].z = input->shear[(e*3)+2];
    numneigh[i]++;

#ifndef NEWTON_THIRD
    assert(numneigh[j] < nslot);
    idx = (j*nslot) + numneigh[j];
    insert_particle(input, neigh, idx, i);
    shear[idx].x = input->shear[(e*3)  ];
    shear[idx].y = input->shear[(e*3)+1];
    shear[idx].z = input->shear[(e*3)+2];
    numneigh[j]++;
#endif
  }

  const int numneigh_size = input->nnode*nslot*sizeof(int);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_numneigh, numneigh_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_numneigh, numneigh, numneigh_size, hipMemcpyHostToDevice));

  const int neigh_size = input->nnode*nslot*sizeof(struct particle);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_neigh, neigh_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_neigh, neigh, neigh_size, hipMemcpyHostToDevice));

  const int shear_size = input->nnode*nslot*sizeof(double3);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_shear, shear_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_shear, shear, shear_size, hipMemcpyHostToDevice));
}

void run(struct params *input, int num_iter) {
  one_time.push_back(SimpleTimer("hertz_constants"));
  one_time.back().start();
  setup_hertz_constants();
  one_time.back().stop_and_add_to_total();

  one_time.push_back(SimpleTimer("build_particle_aos"));
  struct particle *d_particle_aos = NULL;
  one_time.back().start();
  build_particle_aos(input, d_particle_aos);
  one_time.back().stop_and_add_to_total();
  assert(d_particle_aos);

  one_time.push_back(SimpleTimer("build_neigh_list"));
  int *d_numneigh = NULL;
  struct particle *d_neigh = NULL;
  double3 *d_shear = NULL;
  one_time.back().start();
  build_neighbor_list(NSLOT, input, d_numneigh, d_neigh, d_shear);
  one_time.back().stop_and_add_to_total();
  assert(d_numneigh);
  assert(d_neigh);
  assert(d_shear);

  one_time.push_back(SimpleTimer("malloc_force_torque"));
  one_time.back().start();
  double *d_force;
  const int force_size = input->nnode * 3 * sizeof(double);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_force, force_size));

  double *d_torque;
  const int torque_size = input->nnode * 3 * sizeof(double);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_torque, torque_size));
  one_time.back().stop_and_add_to_total();

  per_iter.push_back(SimpleTimer("compute_kernel"));
  for (int run=0; run<num_iter; run++) {
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_force, input->force, force_size, hipMemcpyHostToDevice));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_torque, input->torque, torque_size, hipMemcpyHostToDevice));
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Pre-compute-kernel error: %s.\n", hipGetErrorString(err));
      exit(1);
    }
    const int blockSize = 128;
    dim3 gridSize((input->nnode / blockSize)+1);
    per_iter[0].start();
    compute_kernel_tpa<<<gridSize, blockSize>>>(
      input->nnode, d_particle_aos, d_numneigh, d_neigh, 
      d_shear, d_force, d_torque);
    hipDeviceSynchronize();
    per_iter[0].stop_and_add_to_total();
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Post-compute-kernel error: %s.\n", hipGetErrorString(err));
      exit(1);
    }

#if 0
    double *force_result = new double[input->nnode*3];
    double *torque_result = new double[input->nnode*3];
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(force_result, d_force, force_size, hipMemcpyDeviceToHost));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(torque_result, d_torque, torque_size, hipMemcpyDeviceToHost));
    for (int n=0; n<input->nnode; n++) {
      const double epsilon=0.00001;

      std::stringstream out;
      out << "force[" << n << "]";
      check_result_vector(
          out.str().c_str(),
          &input->expected_force[(n*3)], &force_result[(n*3)], epsilon, false, false);
      out.str("");

      out << "torque[" << n << "]";
      check_result_vector(
          out.str().c_str(),
          &input->expected_torque[(n*3)], &torque_result[(n*3)], epsilon, false, false);
    }
#endif
  }

  hipFree(d_particle_aos);
  hipFree(d_numneigh);
  hipFree(d_neigh);
  hipFree(d_shear);
  hipFree(d_force);
  hipFree(d_torque);
}

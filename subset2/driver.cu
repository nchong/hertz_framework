#include "hip/hip_runtime.h"
#include "hertz_constants.h"
#include "hertz_cudaneighlist.h"
#include "pair_interaction.h"
#include "framework.h"
#include "thrust/scan.h"

#ifdef TRACE
#warning TRACE enabled: timing will not be accurate
#include "cuPrintf.cu"
#endif

#ifndef MAX_GRID_DIM
#error You need to #define MAX_GRID_DIM (see Makefile.config)
#endif

dim3 get_grid(int nelements, int block_size=BLOCK_SIZE) {
  int nx = (nelements + block_size - 1) / block_size;
  if (nx < MAX_GRID_DIM) {
    return dim3(nx, 1, 1);
  }
  int ny = (nx + MAX_GRID_DIM - 1) / MAX_GRID_DIM;
  if (ny < MAX_GRID_DIM) {
    return dim3(MAX_GRID_DIM, ny, 1);
  }
  assert(false);
}

__device__ int get_gid() {
  return threadIdx.x + (blockIdx.x * blockDim.x) + (blockIdx.y * blockDim.x * gridDim.x);
}

// --------------------------------------------------------------------------
// UNPACK PER-PARTICLE DATA
// --------------------------------------------------------------------------
__global__ void unpack_ro_data(
  int K,
                  int    *valid,
                  int    *dati,    int    *datj,
  double *radius, double *radiusi, double *radiusj,
  double *mass,   double *massi,   double *massj,
  int    *type,   int    *typei,   int    *typej
) {
  int gid = get_gid();
  if (gid < K && valid[gid]) {
    int i = dati[gid]; int j = datj[gid];
    radiusi[gid] = radius[i]; radiusj[gid] = radius[j];
    massi[gid]   = mass[i];   massj[gid]   = mass[j];
    typei[gid]   = type[i];   typej[gid]   = type[j];
  }
}

__global__ void test(
  //inputs
  int K,
  int *valid,
  int *dati, int *datj,
  double *x,
  double *radiusi, double *radiusj,
  //output
  int *filter, double *del, double *shear
) {
  int gid = get_gid();
  if (gid < K && valid[gid]) {
    int i = dati[gid];
    int j = datj[gid];
    // del is the vector from j to i
    double delx = x[(i*3)+0] - x[(j*3)+0];
    double dely = x[(i*3)+1] - x[(j*3)+1];
    double delz = x[(i*3)+2] - x[(j*3)+2];

    double rsq = delx*delx + dely*dely + delz*delz;
    double radsum = radiusi[gid] + radiusj[gid];
    if (rsq < radsum*radsum) {
      filter[gid] = 1;
      del[(gid*3)+0] = delx;
      del[(gid*3)+1] = dely;
      del[(gid*3)+2] = delz;
    } else {
      shear[(gid*3)+0] = 0.0;
      shear[(gid*3)+1] = 0.0;
      shear[(gid*3)+2] = 0.0;
    }
  }
}

__global__ void mksubset(
  int K,
  int *filter,
  int *offset,
  //output
  int *hit
) {
  int gid = get_gid();
  if (gid < K && filter[gid]) {
    hit[offset[gid]] = gid;
  }
}

__global__ void compute(
  //inputs
  int NHIT,
  int *hit,
  int    *dati,    int    *datj,
  double *del,
  double *v,
  double *omega,
  double *radiusi, double *radiusj,
  double *massi,   double *massj,
  int    *typei,   int    *typej,
  //inouts
  double *fdelta,
  double *tdeltai, double *tdeltaj,
  double *shear
) {
  int gid = get_gid();
  if (gid < NHIT) {
    int idx = hit[gid];
    int i = dati[idx];
    int j = datj[idx];
    pair_interaction(
#ifdef TRACE
      i, j,
#endif
      &del[idx*3],
      &v[i*3],        &v[j*3],
      &omega[i*3],    &omega[j*3],
      radiusi[idx],   radiusj[idx],
      massi[idx],     massj[idx],
      typei[idx],     typej[idx],
      &shear[idx*3],
      &fdelta[idx*3], /*fdeltaj is*/NULL,
      &tdeltai[idx*3], &tdeltaj[idx*3]
    );
  }
}

__global__ void collect(
  //inputs
  int N,
  double *fdelta,
  double *tdeltai, double *tdeltaj,
            int *off, int *len,
#if HALFNL
  int *tad, int *ffo, int *nel,
#endif
  //inouts
  double *force,
  double *torque
) {
  int gid = get_gid();

  double fsum[3] = {0,0,0};
  double tsum[3] = {0,0,0};
  if (gid < N) {
    int offset = off[gid];
    for (int k=0; k<len[gid]; k++) {
      int idx = offset+k;
      fsum[0] += fdelta[(idx*3)+0];
      fsum[1] += fdelta[(idx*3)+1];
      fsum[2] += fdelta[(idx*3)+2];
      tsum[0] += tdeltai[(idx*3)+0];
      tsum[1] += tdeltai[(idx*3)+1];
      tsum[2] += tdeltai[(idx*3)+2];
    }
#if HALFNL
    offset = ffo[gid];
    for (int k=0; k<nel[gid]; k++) {
      int idx = tad[offset+k];
      fsum[0] -= fdelta[(idx*3)+0];
      fsum[1] -= fdelta[(idx*3)+1];
      fsum[2] -= fdelta[(idx*3)+2];
      tsum[0] += tdeltaj[(idx*3)+0];
      tsum[1] += tdeltaj[(idx*3)+1];
      tsum[2] += tdeltaj[(idx*3)+2];
    }
#endif
    force[(gid*3)]    += fsum[0];
    force[(gid*3)+1]  += fsum[1];
    force[(gid*3)+2]  += fsum[2];
    torque[(gid*3)]   += tsum[0];
    torque[(gid*3)+1] += tsum[1];
    torque[(gid*3)+2] += tsum[2];
  }
}

using namespace std;

// DEVICE STRUCTURES
// INPUTS
// packed         // unpacked(i)     // unpacked(j)
double *d_del;
double *d_x;//    double *d_xi;      double *d_xj;        // ] reload
double *d_v;//    double *d_vi;      double *d_vj;        // ]
double *d_omega;//double *d_omegai;  double *d_omegaj;    // ]
double *d_radius; double *d_radiusi; double *d_radiusj;   // ] ro
double *d_mass;   double *d_massi;   double *d_massj;     // ]
int    *d_type;   int    *d_typei;   int *d_typej;        // ]
// OUTPUTS
// packed         // unpacked(i)     // unpacked(j)
double *d_force;  double *d_fdelta;
double *d_torque; double *d_tdeltai; double *d_tdeltaj;
//                        d_shear in d_nl
// SUBSET
int *d_filter;
int *d_offset;
int *d_hit;

void no_cuda_error(const char *errmsg) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("[ERROR] %s\n", errmsg);
    printf("[ERROR] %d: %s\n", err, hipGetErrorString(err));
    size_t free; size_t total;
    if (hipMemGetInfo(&free, &total) == hipSuccess) {
      printf("[ERROR] mem free=%zubytes total=%zubytes\n", free, total);
    }
    exit(1);
  }
}

#define NLEN(type,arity) (nparticles*arity*sizeof(type))
#define KLEN(type,arity) (nneighbors*arity*sizeof(type))
void init_dev_structures(int nparticles, int nneighbors) {
  //packed
  hipMalloc((void **)&d_x,       NLEN(double,3));
  hipMalloc((void **)&d_v,       NLEN(double,3));
  hipMalloc((void **)&d_omega,   NLEN(double,3));
  hipMalloc((void **)&d_radius,  NLEN(double,1));
  hipMalloc((void **)&d_mass,    NLEN(double,1));
  hipMalloc((void **)&d_type,    NLEN(int,   1));

  //unpacked(i)
  hipMalloc((void **)&d_del,     KLEN(double,3));
  hipMalloc((void **)&d_radiusi, KLEN(double,1));
  hipMalloc((void **)&d_massi,   KLEN(double,1));
  hipMalloc((void **)&d_typei,   KLEN(int   ,1));

  //unpacked(j)
  hipMalloc((void **)&d_radiusj, KLEN(double,1));
  hipMalloc((void **)&d_massj,   KLEN(double,1));
  hipMalloc((void **)&d_typej,   KLEN(int   ,1));

  //outputs
  hipMalloc((void **)&d_force,   NLEN(double,3));
  hipMalloc((void **)&d_torque,  NLEN(double,3));
  hipMalloc((void **)&d_fdelta,  KLEN(double,3));
  hipMalloc((void **)&d_tdeltai, KLEN(double,3));
  hipMalloc((void **)&d_tdeltaj, KLEN(double,3));

  //subset
  hipMalloc((void **)&d_filter, KLEN(int,1));
  hipMalloc((void **)&d_offset, KLEN(int,1));
  hipMalloc((void **)&d_hit,    KLEN(int,1));
}

void free_dev_structures() {
  //packed
  hipFree(d_x);
  hipFree(d_v);
  hipFree(d_omega);
  hipFree(d_radius);
  hipFree(d_mass);
  hipFree(d_type);

  //unpacked(i)
  hipFree(d_del);
  hipFree(d_radiusi);
  hipFree(d_massi);
  hipFree(d_typei);

  //unpacked(j)
  hipFree(d_radiusj);
  hipFree(d_massj);
  hipFree(d_typej);

  //outputs
  hipFree(d_force);
  hipFree(d_torque);
  hipFree(d_fdelta);
  hipFree(d_tdeltai);
  hipFree(d_tdeltaj);

  //subset
  hipFree(d_filter);
  hipFree(d_offset);
  hipFree(d_hit);
}

void run(struct params *input, int num_iter) {
  NeighListLike *nl = new NeighListLike(input);

  int block_size = BLOCK_SIZE;
  int nparticles = input->nnode;
  dim3 tpa_grid_size = get_grid(nparticles);
  int nneighbors = nl->maxpage * nl->pgsize;
  dim3 tpn_grid_size = get_grid(nneighbors);
#if DEBUG
  printf("block_size = %d\n", block_size);
  printf("nparticles = %d\n", nparticles);
  printf("nneighbors = %d -> %d (maxpage=%d, pgsize=%d)\n",
    input->nedge, nneighbors, nl->maxpage, nl->pgsize);
  printf("tpa_grid   = { %d, %d, %d }\n",
    tpa_grid_size.x, tpa_grid_size.y, tpa_grid_size.z);
  printf("tpn_grid   = { %d, %d, %d }\n",
    tpn_grid_size.x, tpn_grid_size.y, tpn_grid_size.z);
#endif

  //ONE-TIME COSTS
  one_time.push_back(SimpleTimer("hertz_consts"));
  one_time.back().start();
  setup_hertz_constants(input);
  one_time.back().stop_and_add_to_total();
  no_cuda_error("hertz_constants");

  one_time.push_back(SimpleTimer("init_nl"));
  one_time.back().start();
  HertzCudaNeighList *d_nl = new HertzCudaNeighList(
    block_size,
    input->nnode,
    nl->maxpage, nl->pgsize);
  one_time.back().stop_and_add_to_total();
  no_cuda_error("init_nl");

  one_time.push_back(SimpleTimer("malloc"));
  one_time.back().start();
  init_dev_structures(nparticles, nneighbors);
  one_time.back().stop_and_add_to_total();
  no_cuda_error("init_dev_structures");

  one_time.push_back(SimpleTimer("memcpy"));
  one_time.back().start();
  hipMemcpy(d_force,  input->force,  NLEN(double,3), hipMemcpyHostToDevice);
  hipMemcpy(d_torque, input->torque, NLEN(double,3), hipMemcpyHostToDevice);
  one_time.back().stop_and_add_to_total();
  no_cuda_error("memcpy");

  //NL-REFRESH COSTS
  nl_refresh.push_back(SimpleTimer("nl_reload"));
  nl_refresh.back().start();
  d_nl->reload(
    nl->numneigh,
    nl->firstneigh,
    nl->pages,
    nl->maxpage,
    nl->dpages,
    nl->tpages);
  nl_refresh.back().stop_and_add_to_total();
  no_cuda_error("nl_reload");

  nl_refresh.push_back(SimpleTimer("memcpy_unpack"));
  nl_refresh.back().start();
  hipMemcpy(d_radius, input->radius, NLEN(double,1), hipMemcpyHostToDevice);
  hipMemcpy(d_mass,   input->mass,   NLEN(double,1), hipMemcpyHostToDevice);
  hipMemcpy(d_type,   input->type,   NLEN(int,1),    hipMemcpyHostToDevice);
  nl_refresh.back().stop_and_add_to_total();
  no_cuda_error("memcpy_unpack");

  nl_refresh.push_back(SimpleTimer("unpack_ro"));
  nl_refresh.back().start();
  unpack_ro_data<<<tpn_grid_size, block_size>>>(
    nneighbors,
              d_nl->d_valid,
              d_nl->d_dati,  d_nl->d_neighidx,
    d_radius, d_radiusi,     d_radiusj,
    d_mass,   d_massi,       d_massj,
    d_type,   d_typei,       d_typej
  );
  hipDeviceSynchronize();
  nl_refresh.back().stop_and_add_to_total();
  no_cuda_error("unpack_ro");

  // PER-ITER COSTS
  per_iter.push_back(SimpleTimer("memcpy_reload"));
  per_iter.push_back(SimpleTimer("unpack_reload"));
  per_iter.push_back(SimpleTimer("memset_delta"));
  per_iter.push_back(SimpleTimer("compute"));
  per_iter.push_back(SimpleTimer("collect"));
  per_iter.push_back(SimpleTimer("memcpy_results"));
  per_iter.push_back(SimpleTimer("mksubset"));
  per_iter.push_back(SimpleTimer("anticompute"));
  for (int i=0; i<(int)per_iter.size(); i++) {
    per_iter_timings.push_back(vector<double>(num_iter));
  }

  double *force  = new double[nparticles*3];
  double *torque = new double[nparticles*3];
  for (int run=0; run<num_iter; run++) {
    //make copies
    nl->restore();
    d_nl->load_shear(nl->dpages);
    no_cuda_error("make_copies");

    end_to_end.start();

    //load data onto device
    per_iter[0].start();
    hipMemcpy(d_x,      input->x,      NLEN(double,3), hipMemcpyHostToDevice);
    hipMemcpy(d_v,      input->v,      NLEN(double,3), hipMemcpyHostToDevice);
    hipMemcpy(d_omega,  input->omega,  NLEN(double,3), hipMemcpyHostToDevice);
    hipMemcpy(d_force,  input->force,  NLEN(double,3), hipMemcpyHostToDevice);
    hipMemcpy(d_torque, input->torque, NLEN(double,3), hipMemcpyHostToDevice);
    double d0 = per_iter[0].stop_and_add_to_total();
    per_iter_timings[0][run] = d0;
    no_cuda_error("memcpy_reload");

    per_iter[2].start();
    hipMemset(d_fdelta,  0, KLEN(double,3));
    hipMemset(d_tdeltai, 0, KLEN(double,3));
    hipMemset(d_tdeltaj, 0, KLEN(double,3));
    double d2 = per_iter[2].stop_and_add_to_total();
    per_iter_timings[2][run] = d2;
    no_cuda_error("memset_delta");

    //make subset
    per_iter[6].start();
    hipMemset(d_filter, 0, KLEN(int,1));
    test<<<tpn_grid_size, block_size>>>(
      //inputs
      nneighbors,
      d_nl->d_valid,
      d_nl->d_dati,  d_nl->d_neighidx,
      d_x,
      d_radiusi,     d_radiusj,
      //outputs
      d_filter, d_del, d_nl->d_shear);
    thrust::device_ptr<int> thrust_filter(d_filter);
    thrust::device_ptr<int> thrust_offset(d_offset);
    thrust::exclusive_scan(thrust_filter, thrust_filter + nneighbors, thrust_offset);
    mksubset<<<tpn_grid_size, block_size>>>(
      //inputs
      nneighbors,
      d_filter,
      d_offset,
      //output
      d_hit);
    int nhit;
    hipMemcpy(&nhit, &(d_offset[nneighbors-1]), sizeof(int), hipMemcpyDeviceToHost);
    double d6 = per_iter[6].stop_and_add_to_total();
    per_iter_timings[6][run] = d6;
    dim3 nhit_grid_size = get_grid(nhit);
    no_cuda_error("mksubset");
#if DEBUG
  printf("nhit        = %d\n", nhit);
  printf("nhit_grid   = { %d, %d, %d }\n",
    nhit_grid_size.x, nhit_grid_size.y, nhit_grid_size.z);
#endif

    per_iter[3].start();
#ifdef TRACE
    cudaPrintfInit();
#endif
    compute<<<nhit_grid_size, block_size>>>(
      nhit,
      d_hit,
      d_nl->d_dati,  d_nl->d_neighidx,
      d_del,
      d_v,
      d_omega,
      d_radiusi,     d_radiusj,
      d_massi,       d_massj,
      d_typei,       d_typej,
      //outputs
      d_fdelta,
      d_tdeltai,     d_tdeltaj,
      d_nl->d_shear
    );
    hipDeviceSynchronize();
    double d3 = per_iter[3].stop_and_add_to_total();
    per_iter_timings[3][run] = d3;
    no_cuda_error("compute");
#ifdef TRACE
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
#endif

    per_iter[4].start();
    collect<<<tpa_grid_size, block_size>>>(
      nparticles,
      d_fdelta,
      d_tdeltai, d_tdeltaj,
                        d_nl->d_offset, d_nl->d_numneigh,
#if HALFNL
      d_nl->d_tad,      d_nl->d_ffo,    d_nl->d_nel,
#endif
      d_force,
      d_torque);
    hipDeviceSynchronize();
    double d4 = per_iter[4].stop_and_add_to_total();
    per_iter_timings[4][run] = d4;
    no_cuda_error("collect");

    //offload data from device
    //(see note on shear history below)
    per_iter[5].start();
    hipMemcpy(force,  d_force,  NLEN(double,3), hipMemcpyDeviceToHost);
    hipMemcpy(torque, d_torque, NLEN(double,3), hipMemcpyDeviceToHost);
    double d5 = per_iter[5].stop_and_add_to_total();
    per_iter_timings[5][run] = d5;
    no_cuda_error("memcpy_results");

    double dend = end_to_end.stop_and_add_to_total();
    end_to_end_timings.push_back(dend);

    //NB: we assume that shear history is *not* required from the device
    //so this cost is not included in "memcpy_results"
    d_nl->unload_shear(nl->dpages);
    check_result(input, nl, force, torque, nl->firstdouble,
      /*threshold=*/0.5,
      /*verbose=*/false,
      /*die_on_flag=*/true);
  }
  delete[] force;
  delete[] torque;
  free_dev_structures();
  no_cuda_error("free_dev_structures");
}


#include "hip/hip_runtime.h"
#include "hertz_constants.h"
#include "hertz_cudaneighlist.h"
#include "pair_interaction.h"
#include "framework.h"

#ifdef TRACE
#warning TRACE enabled: timing will not be accurate
#include "cuPrintf.cu"
#endif

#ifndef MAX_GRID_DIM
#error You need to #define MAX_GRID_DIM (see Makefile.config)
#endif

__device__ int get_gid() {
  return threadIdx.x + (blockIdx.x * blockDim.x) + (blockIdx.y * blockDim.x * gridDim.x);
}

// --------------------------------------------------------------------------
// UNPACK PER-PARTICLE DATA
// --------------------------------------------------------------------------
__global__ void unpack_ro_data(
  int K,
                  int    *valid,
                  int    *dati,    int    *datj,
  double *radius, double *radiusi, double *radiusj,
  double *mass,   double *massi,   double *massj,
  int    *type,   int    *typei,   int    *typej
) {
  int gid = get_gid();
  if (gid < K && valid[gid]) {
    int i = dati[gid]; int j = datj[gid];
    radiusi[gid] = radius[i]; radiusj[gid] = radius[j];
    massi[gid]   = mass[i];   massj[gid]   = mass[j];
    typei[gid]   = type[i];   typej[gid]   = type[j];
  }
}

__global__ void unpack_reload_data(
  int K,
                  int    *valid,
                  int    *dati,    int    *datj,
  double *x,      double *xi,      double *xj,
  double *v,      double *vi,      double *vj,
  double *omega,  double *omegai,  double *omegaj
) {
  int gid = get_gid();
  if (gid < K && valid[gid]) {
    int i = dati[gid]; int j = datj[gid];
    xi[(gid*3)+0] = x[(i*3)+0]; xj[(gid*3)+0] = x[(j*3)+0];
    xi[(gid*3)+1] = x[(i*3)+1]; xj[(gid*3)+1] = x[(j*3)+1];
    xi[(gid*3)+2] = x[(i*3)+2]; xj[(gid*3)+2] = x[(j*3)+2];
    vi[(gid*3)+0] = v[(i*3)+0]; vj[(gid*3)+0] = v[(j*3)+0];
    vi[(gid*3)+1] = v[(i*3)+1]; vj[(gid*3)+1] = v[(j*3)+1];
    vi[(gid*3)+2] = v[(i*3)+2]; vj[(gid*3)+2] = v[(j*3)+2];
    omegai[(gid*3)+0] = omega[(i*3)+0]; omegaj[(gid*3)+0] = omega[(j*3)+0];
    omegai[(gid*3)+1] = omega[(i*3)+1]; omegaj[(gid*3)+1] = omega[(j*3)+1];
    omegai[(gid*3)+2] = omega[(i*3)+2]; omegaj[(gid*3)+2] = omega[(j*3)+2];
  }
}

__global__ void compute(
  //inputs
  int K,
  int    *valid,
#ifdef TRACE
  int    *dati,    int    *datj,
#endif
  double *xi,      double *xj,
  double *vi,      double *vj,
  double *omegai,  double *omegaj,
  double *radiusi, double *radiusj,
  double *massi,   double *massj,
  int    *typei,   int    *typej,
  //inouts
  double *fdelta,
  double *tdeltai, double *tdeltaj,
  double *shear
) {
  int gid = get_gid();
  if (gid < K && valid[gid]) {
    pair_interaction(
#ifdef TRACE
      dati[gid], datj[gid],
#endif
      &xi[gid*3],     &xj[gid*3],
      &vi[gid*3],     &vj[gid*3],
      &omegai[gid*3], &omegaj[gid*3],
      radiusi[gid],   radiusj[gid],
      massi[gid],     massj[gid],
      typei[gid],     typej[gid],
      &shear[gid*3],
      &fdelta[gid*3], /*fdeltaj is*/NULL,
      &tdeltai[gid*3], &tdeltaj[gid*3]
    );
  }
}

__global__ void collect(
  //inputs
  int N,
  double *fdelta,
  double *tdeltai, double *tdeltaj,
            int *off, int *len,
#if HALFNL
  int *tad, int *ffo, int *nel,
#endif
  //inouts
  double *force,
  double *torque
) {
  int gid = get_gid();

  double fsum[3] = {0,0,0};
  double tsum[3] = {0,0,0};
  if (gid < N) {
    int offset = off[gid];
    for (int k=0; k<len[gid]; k++) {
      int idx = offset+k;
      fsum[0] += fdelta[(idx*3)+0];
      fsum[1] += fdelta[(idx*3)+1];
      fsum[2] += fdelta[(idx*3)+2];
      tsum[0] += tdeltai[(idx*3)+0];
      tsum[1] += tdeltai[(idx*3)+1];
      tsum[2] += tdeltai[(idx*3)+2];
    }
#if HALFNL
    offset = ffo[gid];
    for (int k=0; k<nel[gid]; k++) {
      int idx = tad[offset+k];
      fsum[0] -= fdelta[(idx*3)+0];
      fsum[1] -= fdelta[(idx*3)+1];
      fsum[2] -= fdelta[(idx*3)+2];
      tsum[0] += tdeltaj[(idx*3)+0];
      tsum[1] += tdeltaj[(idx*3)+1];
      tsum[2] += tdeltaj[(idx*3)+2];
    }
    force[(gid*3)]    += fsum[0];
    force[(gid*3)+1]  += fsum[1];
    force[(gid*3)+2]  += fsum[2];
    torque[(gid*3)]   += tsum[0];
    torque[(gid*3)+1] += tsum[1];
    torque[(gid*3)+2] += tsum[2];
#endif
  }
}

using namespace std;

// DEVICE STRUCTURES
// INPUTS
// packed         // unpacked(i)     // unpacked(j)
double *d_x;      double *d_xi;      double *d_xj;        // ] reload
double *d_v;      double *d_vi;      double *d_vj;        // ]
double *d_omega;  double *d_omegai;  double *d_omegaj;    // ]
double *d_radius; double *d_radiusi; double *d_radiusj;   // ] ro
double *d_mass;   double *d_massi;   double *d_massj;     // ]
int    *d_type;   int    *d_typei;   int *d_typej;        // ]
// OUTPUTS
// packed         // unpacked(i)     // unpacked(j)
double *d_force;  double *d_fdelta;
double *d_torque; double *d_tdeltai; double *d_tdeltaj;
//                        d_shear in d_nl

void no_cuda_error(const char *errmsg) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("[ERROR] %s\n", errmsg);
    printf("[ERROR] %d: %s\n", err, hipGetErrorString(err));
    size_t free; size_t total;
    if (hipMemGetInfo(&free, &total) == hipSuccess) {
      printf("[ERROR] mem free=%zubytes total=%zubytes\n", free, total);
    }
    exit(1);
  }
}

#define NLEN(type,arity) (nparticles*arity*sizeof(type))
#define KLEN(type,arity) (nneighbors*arity*sizeof(type))
void init_dev_structures(int nparticles, int nneighbors) {
  //packed
  hipMalloc((void **)&d_x,       NLEN(double,3));
  hipMalloc((void **)&d_v,       NLEN(double,3));
  hipMalloc((void **)&d_omega,   NLEN(double,3));
  hipMalloc((void **)&d_radius,  NLEN(double,1));
  hipMalloc((void **)&d_mass,    NLEN(double,1));
  hipMalloc((void **)&d_type,    NLEN(int,   1));

  //unpacked(i)
  hipMalloc((void **)&d_xi,      KLEN(double,3));
  hipMalloc((void **)&d_vi,      KLEN(double,3));
  hipMalloc((void **)&d_omegai,  KLEN(double,3));
  hipMalloc((void **)&d_radiusi, KLEN(double,1));
  hipMalloc((void **)&d_massi,   KLEN(double,1));
  hipMalloc((void **)&d_typei,   KLEN(int   ,1));

  //unpacked(j)
  hipMalloc((void **)&d_xj,      KLEN(double,3));
  hipMalloc((void **)&d_vj,      KLEN(double,3));
  hipMalloc((void **)&d_omegaj,  KLEN(double,3));
  hipMalloc((void **)&d_radiusj, KLEN(double,1));
  hipMalloc((void **)&d_massj,   KLEN(double,1));
  hipMalloc((void **)&d_typej,   KLEN(int   ,1));

  //outputs
  hipMalloc((void **)&d_force,   NLEN(double,3));
  hipMalloc((void **)&d_torque,  NLEN(double,3));
  hipMalloc((void **)&d_fdelta,  KLEN(double,3));
  hipMalloc((void **)&d_tdeltai, KLEN(double,3));
  hipMalloc((void **)&d_tdeltaj, KLEN(double,3));
}

void free_dev_structures() {
  //packed
  hipFree(d_x);
  hipFree(d_v);
  hipFree(d_omega);
  hipFree(d_radius);
  hipFree(d_mass);
  hipFree(d_type);

  //unpacked(i)
  hipFree(d_xi);
  hipFree(d_vi);
  hipFree(d_omegai);
  hipFree(d_radiusi);
  hipFree(d_massi);
  hipFree(d_typei);

  //unpacked(j)
  hipFree(d_xj);
  hipFree(d_vj);
  hipFree(d_omegaj);
  hipFree(d_radiusj);
  hipFree(d_massj);
  hipFree(d_typej);

  //outputs
  hipFree(d_force);
  hipFree(d_torque);
  hipFree(d_fdelta);
  hipFree(d_tdeltai);
  hipFree(d_tdeltaj);
}

void run(struct params *input, int num_iter) {
  NeighListLike *nl = new NeighListLike(input);

  int block_size = BLOCK_SIZE;
  int nparticles = input->nnode;
  dim3 tpa_grid_size(
    min(nparticles/block_size, MAX_GRID_DIM),
    max((int)ceil(((float)nparticles/block_size)/MAX_GRID_DIM), 1));
  int nneighbors = nl->maxpage * nl->pgsize;
  dim3 tpn_grid_size(
    min(nneighbors/block_size, MAX_GRID_DIM),
    max((int)ceil(((float)nneighbors/block_size)/MAX_GRID_DIM), 1));
#if DEBUG
  printf("block_size = %d\n", block_size);
  printf("nparticles = %d\n", nparticles);
  printf("nneighbors = %d -> %d (maxpage=%d, pgsize=%d)\n",
    input->nedge, nneighbors, nl->maxpage, nl->pgsize);
  printf("tpa_grid   = { %d, %d, %d }\n",
    tpa_grid_size.x, tpa_grid_size.y, tpa_grid_size.z);
  printf("tpn_grid   = { %d, %d, %d }\n",
    tpn_grid_size.x, tpn_grid_size.y, tpn_grid_size.z);
#endif

  //ONE-TIME COSTS
  one_time.push_back(SimpleTimer("hertz_consts"));
  one_time.back().start();
  setup_hertz_constants(input);
  one_time.back().stop_and_add_to_total();
  no_cuda_error("hertz_constants");

  one_time.push_back(SimpleTimer("init_nl"));
  one_time.back().start();
  HertzCudaNeighList *d_nl = new HertzCudaNeighList(
    block_size,
    input->nnode,
    nl->maxpage, nl->pgsize);
  one_time.back().stop_and_add_to_total();
  no_cuda_error("init_nl");

  one_time.push_back(SimpleTimer("malloc"));
  one_time.back().start();
  init_dev_structures(nparticles, nneighbors);
  one_time.back().stop_and_add_to_total();
  no_cuda_error("init_dev_structures");

  one_time.push_back(SimpleTimer("memcpy"));
  one_time.back().start();
  hipMemcpy(d_force,  input->force,  NLEN(double,3), hipMemcpyHostToDevice);
  hipMemcpy(d_torque, input->torque, NLEN(double,3), hipMemcpyHostToDevice);
  one_time.back().stop_and_add_to_total();
  no_cuda_error("memcpy");

  //NL-REFRESH COSTS
  nl_refresh.push_back(SimpleTimer("nl_reload"));
  nl_refresh.back().start();
  d_nl->reload(
    nl->numneigh,
    nl->firstneigh,
    nl->pages,
    nl->maxpage,
    nl->dpages,
    nl->tpages);
  nl_refresh.back().stop_and_add_to_total();
  no_cuda_error("nl_reload");

  nl_refresh.push_back(SimpleTimer("memcpy_unpack"));
  nl_refresh.back().start();
  hipMemcpy(d_radius, input->radius, NLEN(double,1), hipMemcpyHostToDevice);
  hipMemcpy(d_mass,   input->mass,   NLEN(double,1), hipMemcpyHostToDevice);
  hipMemcpy(d_type,   input->type,   NLEN(int,1),    hipMemcpyHostToDevice);
  nl_refresh.back().stop_and_add_to_total();
  no_cuda_error("memcpy_unpack");

  nl_refresh.push_back(SimpleTimer("unpack_ro"));
  nl_refresh.back().start();
  unpack_ro_data<<<tpn_grid_size, block_size>>>(
    nneighbors,
              d_nl->d_valid,
              d_nl->d_dati,  d_nl->d_neighidx,
    d_radius, d_radiusi,     d_radiusj,
    d_mass,   d_massi,       d_massj,
    d_type,   d_typei,       d_typej
  );
  hipDeviceSynchronize();
  nl_refresh.back().stop_and_add_to_total();
  no_cuda_error("unpack_ro");

  // PER-ITER COSTS
  per_iter.push_back(SimpleTimer("memcpy_reload"));
  per_iter.push_back(SimpleTimer("unpack_reload"));
  per_iter.push_back(SimpleTimer("compute"));
  per_iter.push_back(SimpleTimer("collect"));
  per_iter.push_back(SimpleTimer("memcpy_results"));
  per_iter_timings.push_back(vector<double>(num_iter));
  per_iter_timings.push_back(vector<double>(num_iter));
  per_iter_timings.push_back(vector<double>(num_iter));
  per_iter_timings.push_back(vector<double>(num_iter));
  per_iter_timings.push_back(vector<double>(num_iter));

  double *force  = new double[nparticles*3];
  double *torque = new double[nparticles*3];
  for (int run=0; run<num_iter; run++) {
    //make copies
    hipMemcpy(d_force,  input->force,  NLEN(double,3), hipMemcpyHostToDevice);
    hipMemcpy(d_torque, input->torque, NLEN(double,3), hipMemcpyHostToDevice);
    hipMemset(d_fdelta,  0, KLEN(double,3));
    hipMemset(d_tdeltai, 0, KLEN(double,3));
    hipMemset(d_tdeltaj, 0, KLEN(double,3));
    nl->restore();
    d_nl->load_shear(nl->dpages);
    no_cuda_error("make_copies");

    end_to_end.start();

    per_iter[0].start();
    hipMemcpy(d_x,     input->x,     NLEN(double,3), hipMemcpyHostToDevice);
    hipMemcpy(d_v,     input->v,     NLEN(double,3), hipMemcpyHostToDevice);
    hipMemcpy(d_omega, input->omega, NLEN(double,3), hipMemcpyHostToDevice);
    double d0 = per_iter[0].stop_and_add_to_total();
    per_iter_timings[0][run] = d0;
    no_cuda_error("memcpy_reload");

    //TODO: check if realloc of unpacked ij data necessary
    per_iter[1].start();
    unpack_reload_data<<<tpn_grid_size, block_size>>>(
      nneighbors,
               d_nl->d_valid,
               d_nl->d_dati,  d_nl->d_neighidx,
      d_x,     d_xi,          d_xj,
      d_v,     d_vi,          d_vj,
      d_omega, d_omegai,      d_omegaj
    );
    hipDeviceSynchronize();
    double d1 = per_iter[1].stop_and_add_to_total();
    per_iter_timings[1][run] = d1;
    no_cuda_error("unpack_reload");

    per_iter[2].start();
#ifdef TRACE
    cudaPrintfInit();
#endif
    compute<<<tpn_grid_size, block_size>>>(
      nneighbors,
      d_nl->d_valid,
#ifdef TRACE
      d_nl->d_dati,  d_nl->d_neighidx,
#endif
      d_xi,          d_xj,
      d_vi,          d_vj,
      d_omegai,      d_omegaj,
      d_radiusi,     d_radiusj,
      d_massi,       d_massj,
      d_typei,       d_typej,
      //outputs
      d_fdelta,
      d_tdeltai,     d_tdeltaj,
      d_nl->d_shear
    );
    hipDeviceSynchronize();
    double d2 = per_iter[2].stop_and_add_to_total();
    per_iter_timings[2][run] = d2;
    no_cuda_error("compute");
#ifdef TRACE
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
#endif

    per_iter[3].start();
    collect<<<tpa_grid_size, block_size>>>(
      nparticles,
      d_fdelta,
      d_tdeltai, d_tdeltaj,
                        d_nl->d_offset, d_nl->d_numneigh,
#if HALFNL
      d_nl->d_tad,      d_nl->d_ffo,    d_nl->d_nel,
#endif
      d_force,
      d_torque);
    hipDeviceSynchronize();
    double d3 = per_iter[3].stop_and_add_to_total();
    per_iter_timings[3][run] = d3;
    no_cuda_error("collect");

    per_iter[4].start();
    hipMemcpy(force,  d_force,  NLEN(double,3), hipMemcpyDeviceToHost);
    hipMemcpy(torque, d_torque, NLEN(double,3), hipMemcpyDeviceToHost);
    d_nl->unload_shear(nl->dpages);
    double d4 = per_iter[4].stop_and_add_to_total();
    per_iter_timings[4][run] = d4;
    no_cuda_error("memcpy_results");

    double d5 = end_to_end.stop_and_add_to_total();
    end_to_end_timings.push_back(d5);

    check_result(input, nl, force, torque, nl->firstdouble,
      /*threshold=*/0.5,
      /*verbose=*/false,
      /*die_on_flag=*/true);
  }
  delete[] force;
  delete[] torque;
  free_dev_structures();
  no_cuda_error("free_dev_structures");
}

